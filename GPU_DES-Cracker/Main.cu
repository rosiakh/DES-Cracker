#include "hip/hip_runtime.h"
#include "DES-Cracker.cuh"

#define THREADS_PER_BLOCK 1024
#define NUM_BLOCKS 32

//print bits in positions from 1 to pos
void print_uint64_t(uint64_t bits, uint32_t pos, std::string label)
{
	std::cout << label << ": ";
	for (int i = 1; i <= pos; ++i)
	{
		std::cout << ((bits >> (64 - i)) & 0x01);
	}
	std::cout << std::endl;
}

void experiment(std::string plaintext_string, std::string key_string)
{
	dim3 threads_per_block(THREADS_PER_BLOCK);
	dim3 num_blocks(NUM_BLOCKS);

	uint64_t plaintext = strtoll(plaintext_string.c_str(), nullptr, 2);
	uint64_t key = strtoll(key_string.c_str(), nullptr, 2);
	uint64_t ciphertext = CPU_namespace::encrypt(plaintext, key);

	uint64_t base_try_key = 0LL;
	uint64_t keys_tried = 0LL;

	//variables for communication with GPU
	bool is_key_found = false, *d_is_key_found;
	uint64_t result_key, *d_result_key;

	hipMalloc(&d_is_key_found, sizeof(bool));
	hipMalloc(&d_result_key, sizeof(uint64_t));
	hipMemset(d_is_key_found, 0, sizeof(bool));
	hipMemset(d_result_key, 0, sizeof(uint64_t));

	std::cout << "CUDA version:\n\n";

	clock_t start;
	double duration;

	start = clock();
	while (true)
	{
		check_keys << <num_blocks, threads_per_block >> >(plaintext, base_try_key, ciphertext, d_result_key, d_is_key_found);
		keys_tried += NUM_BLOCKS*THREADS_PER_BLOCK;
		base_try_key += NUM_BLOCKS*THREADS_PER_BLOCK;

		std::cout << "\rKeys tried: " << keys_tried;

		hipMemcpy(&is_key_found, d_is_key_found, sizeof(bool), hipMemcpyDeviceToHost);
		if (is_key_found)
		{
			hipMemcpy(&result_key, d_result_key, sizeof(uint64_t), hipMemcpyDeviceToHost);
			break;
		}
	}
	duration = (clock() - start) / static_cast<double>(CLOCKS_PER_SEC);
	std::cout << "\rKeys tried: " << ++keys_tried << std::endl;
	std::cout << "Time of brute force attack: " << duration << " s\n";
	print_uint64_t(result_key, 64, "Key found");

	std::cout << std::endl;
}

void experiment_no_permutations(std::string plaintext_string, std::string key_string)
{
	dim3 threads_per_block(THREADS_PER_BLOCK);
	dim3 num_blocks(NUM_BLOCKS);

	uint64_t plaintext = strtoll(plaintext_string.c_str(), nullptr, 2);
	uint64_t key = strtoll(key_string.c_str(), nullptr, 2);
	uint64_t ciphertext = CPU_namespace::encrypt(plaintext, key);

	uint64_t ip = CPU_namespace::permute_IP(plaintext);
	uint64_t no_perms = CPU_namespace::encrypt_no_permutations(ip, key);

	uint64_t base_try_key = 0LL;
	uint64_t keys_tried = 0LL;

	//variables for communication with GPU
	bool is_key_found = false, *d_is_key_found;
	uint64_t result_key, *d_result_key;

	hipMalloc(&d_is_key_found, sizeof(bool));
	hipMalloc(&d_result_key, sizeof(uint64_t));
	hipMemset(d_is_key_found, 0, sizeof(bool));
	hipMemset(d_result_key, 0, sizeof(uint64_t));

	std::cout << "CUDA version (preapplied permutations):\n\n";

	clock_t start;
	double duration;

	start = clock();
	while (true)
	{
		check_keys_no_permutations << <num_blocks, threads_per_block >> >(ip, base_try_key, no_perms, d_result_key, d_is_key_found);
		keys_tried += NUM_BLOCKS*THREADS_PER_BLOCK;
		base_try_key += NUM_BLOCKS*THREADS_PER_BLOCK;

		std::cout << "\rKeys tried: " << keys_tried;

		hipMemcpy(&is_key_found, d_is_key_found, sizeof(bool), hipMemcpyDeviceToHost);
		if (is_key_found)
		{
			hipMemcpy(&result_key, d_result_key, sizeof(uint64_t), hipMemcpyDeviceToHost);
			break;
		}
	}

	duration = (clock() - start) / static_cast<double>(CLOCKS_PER_SEC);
	std::cout << "\rKeys tried: " << ++keys_tried << std::endl;
	std::cout << "Time of brute force attack: " << duration << " s\n";
	print_uint64_t(result_key, 64, "Key found");

	std::cout << std::endl;
}

int main()
{
	std::string plaintext_string = "0000000100100011010001010110011110001001101010111100110111101111";
	std::string key_string = "0000000000000000000000000000000000000000011111110100000000000000";

	experiment(plaintext_string, key_string);
	experiment_no_permutations(plaintext_string, key_string);
}
